//this definition is to define global variable for GPU
#define GLB_GPU

#include "global.h"
#include "global_gpu.h"
#include "gpu.h"
#include "geometry_init.h"

void init_neighbors_gpu() 
{
  #ifdef WITH_GPU
  #ifdef WITH_MPI
    int N = T_EXT*X_EXT*Y_EXT*Z_EXT;
  #else
    int N = T*X*Y*Z;
  #endif

  hipError_t error_id;
  error_id = hipMalloc((void **)&iup_gpu, 4 * N * sizeof(int));
  error(error_id != hipSuccess, 1, "init_neighbors_gpu", hipGetErrorString(error_id));

  error_id = hipMalloc((void **)&idn_gpu, 4 * N * sizeof(int));
  error(error_id != hipSuccess, 1, "init_neighbors_gpu", "Error allocating idn_gpu neighbors array.\n");

  error_id = hipMalloc((void **)&imask_gpu, 8 * N * sizeof(char));
  error(error_id != hipSuccess, 1, "init_neighbors_gpu", "Error allocating imask_gpu lookup table.\n");

  error_id = hipMalloc((void **)&ipt_gpu, (X+2*X_BORDER)*(Y+2*Y_BORDER)*(Z+2*Z_BORDER)*(T+2*T_BORDER)*sizeof(int));
  error(error_id != hipSuccess, 1, "init_neighbors_gpu", "Error allocating ipt_gpu lookup table.\n");

  error_id = hipMemcpy(iup_gpu, iup, 4 * N * sizeof(int), hipMemcpyHostToDevice);
  error(error_id != hipSuccess, 1, "init_neighbors_gpu", "Error copying iup neighbors array to device memory.\n");

  error_id = hipMemcpy(idn_gpu, idn, 4 * N * sizeof(int), hipMemcpyHostToDevice);
  error(error_id != hipSuccess, 1, "init_neighbors_gpu", "Error copying idn neighbors array to device memory.\n");

  error_id = hipMemcpy(imask_gpu, imask, N * sizeof(*imask), hipMemcpyHostToDevice);
  error(error_id != hipSuccess, 1, "init_neighbors_gpu", "Error copying imask lookup table to device memory.\n");

  error_id = hipMemcpy(ipt_gpu, ipt, (X+2*X_BORDER)*(Y+2*Y_BORDER)*(Z+2*Z_BORDER)*(T+2*T_BORDER)*sizeof(int), hipMemcpyHostToDevice);
  error(error_id != hipSuccess, 1, "init_neighbors_gpu", "Error copying ipt to device memory.\n");

  error_id = hipMemcpyToSymbol(HIP_SYMBOL(T_EXT_GPU), &T_EXT, sizeof(int), 0, hipMemcpyHostToDevice);
  error(error_id != hipSuccess, 1, "init_neighbors_gpu", "Error adding T_EXT to global constant memory.\n");

  error_id = hipMemcpyToSymbol(HIP_SYMBOL(X_EXT_GPU), &X_EXT, sizeof(int), 0, hipMemcpyHostToDevice);
  error(error_id != hipSuccess, 1, "init_neighbors_gpu", "Error adding X_EXT to global constant memory.\n");

  error_id = hipMemcpyToSymbol(HIP_SYMBOL(Y_EXT_GPU), &Y_EXT, sizeof(int), 0, hipMemcpyHostToDevice);
  error(error_id != hipSuccess, 1, "init_neighbors_gpu", "Error adding Y_EXT to global constant memory.\n");

  error_id = hipMemcpyToSymbol(HIP_SYMBOL(Z_EXT_GPU), &Z_EXT, sizeof(int), 0, hipMemcpyHostToDevice);
  error(error_id != hipSuccess, 1, "init_neighbors_gpu", "Error adding Z_EXT to global constant memory.\n");
  #endif
}

<<<<<<< HEAD
#undef GLB_GPU

=======
>>>>>>> remotes/upstream/HiRep-CUDA
