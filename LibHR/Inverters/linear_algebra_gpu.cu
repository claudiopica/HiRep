
#include <hip/hip_runtime.h>
/***************************************************************************\
* Copyright (c) 2008, Claudio Pica                                          *
* All rights reserved.                                                      *
\***************************************************************************/
#ifdef WITH_GPU
//This file should not be compiled if !WITH_GPU

#include "inverters.h"
#include "libhr_core.h"
#include "geometry.h"
#include "Utils/generics.h"

// Linear Algebra functions are generic
// They are parametrized over the input types for double/single precision
// The template for GPU is in TMPL/linear_algebra_gpu.cu.tmpl

// double precision 
#define _SPINOR_FIELD_TYPE spinor_field
#define _SPINOR_TYPE suNf_spinor
#define _REAL double
#define _COMPLEX hr_complex
#define _SUFFIX _f
#include "TMPL/linear_algebra_gpu.cu.tmpl"

// single precision 
#define _SPINOR_FIELD_TYPE spinor_field_flt
#define _SPINOR_TYPE suNf_spinor_flt
#define _REAL float
#define _COMPLEX hr_complex_flt
#define _SUFFIX _f_flt
#include "TMPL/linear_algebra_gpu.cu.tmpl"

#endif
